#include "hip/hip_runtime.h"
#include "../common/utils.h"

// 输入矩阵维度
#define INPUT_M 200
#define INPUT_N 700
#define INPUT_L 500
// 线程数和block数
#define THREADS 512
#define BLOCKS 4


__global__ void matrix_transposition_gpu_1d(DATATYPE* a, DATATYPE* b, int m, int n)
{ 
    int tidx = threadIdx.x;
    int bidx = blockIdx.x;
    // 每个block处理矩阵的1行（列）
    while (bidx < m)
    {
        while (tidx < n)
        {
            b[tidx * m + bidx] = a[bidx * n + tidx];
            tidx += blockDim.x;
        }
        bidx += gridDim.x;
    }
}


int main()
{
    size_t size_a = sizeof(DATATYPE) * INPUT_M * INPUT_N;
    size_t size_b = sizeof(DATATYPE) * INPUT_N * INPUT_M;
    DATATYPE* h_a = (DATATYPE*)malloc(size_a);
    DATATYPE* h_b = (DATATYPE*)malloc(size_b);
    // 输入初始化
    for (int i = 0; i < INPUT_M; ++i)
    {
        for (int j = 0; j < INPUT_N; ++j)
        {
            h_a[i * INPUT_N + j] = rand() / (DATATYPE)RAND_MAX;
        }
    }
    // baseline
    DATATYPE* baseline = (DATATYPE*)malloc(size_b);
    matrix_transpose_baseline(h_a, baseline, INPUT_M, INPUT_N);
    // 分配设备上的内存
    DATATYPE* d_a = NULL;
    hipMalloc((void**)&d_a, size_a);
    DATATYPE* d_b = NULL;
    hipMalloc((void**)&d_b, size_b);
    hipError_t err = hipGetLastError();
    if (err != 0)
    {
        printf("error in hipMalloc: %s\n", hipGetErrorString(err));
    }
    // 数据拷贝
    hipMemcpy(d_a, h_a, size_a, hipMemcpyHostToDevice);
    err = hipGetLastError();
    if (err != 0)
    {
        printf("error in hipMemcpy: %s\n", hipGetErrorString(err));
    }
    // 使用grid内所有线程计算
    {
        // 定义启动核函数的参数
        dim3 blocksPerGrid(BLOCKS, 1, 1);
        dim3 threadsPerBlock(THREADS, 1, 1);
        matrix_transposition_gpu_1d<<<blocksPerGrid, threadsPerBlock>>>(
            d_a, d_b, INPUT_M, INPUT_N);
        err = hipGetLastError();
        if (err != 0)
        {
            printf("error in kernel forward: %s\n", hipGetErrorString(err));
        }
        // 拷贝输出数据
        hipMemcpy(h_b, d_b, size_b, hipMemcpyDeviceToHost);
        check_matrix(baseline, h_b, INPUT_N, INPUT_M);
    }
    // 内存释放
    hipFree(d_a);
    hipFree(d_b);
    free(h_a);
    free(h_b);
    return 0;
}
