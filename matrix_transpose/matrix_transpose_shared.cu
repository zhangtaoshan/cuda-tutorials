#include "hip/hip_runtime.h"
#include "../common/utils.h"

// 输入矩阵维度
#define INPUT_M 200
#define INPUT_N 700
// 线程数和block数
#define BLOCK_DIM 16


__global__ void matrix_transposition_gpu_2d_2(DATATYPE* a, DATATYPE* b, int m, int n)
{
    __shared__ DATATYPE tmp[BLOCK_DIM][BLOCK_DIM + 1];
    // put the elements from a to shared memory
    int xIndex = blockIdx.x * BLOCK_DIM + threadIdx.x;
    int yIndex = blockIdx.y * BLOCK_DIM + threadIdx.y;
    if (xIndex < n && yIndex < m)
    {
        // a=(m,n)
        tmp[threadIdx.y][threadIdx.x] = a[yIndex * n + xIndex];
    }
    __syncthreads();
    xIndex = blockIdx.y * BLOCK_DIM + threadIdx.x;
    yIndex = blockIdx.x * BLOCK_DIM + threadIdx.y;
    if (xIndex < m && yIndex < n)
    {
        // b=(n,m)
        b[yIndex * m + xIndex] = tmp[threadIdx.x][threadIdx.y];
    }
}


int main()
{
    size_t size_a = sizeof(DATATYPE) * INPUT_M * INPUT_N;
    size_t size_b = sizeof(DATATYPE) * INPUT_N * INPUT_M;
    DATATYPE* h_a = (DATATYPE*)malloc(size_a);
    DATATYPE* h_b = (DATATYPE*)malloc(size_b);
    // 输入初始化
    for (int i = 0; i < INPUT_M; ++i)
    {
        for (int j = 0; j < INPUT_N; ++j)
        {
            h_a[i * INPUT_N + j] = rand() / (DATATYPE)RAND_MAX;
        }
    }
    // baseline
    DATATYPE* baseline = (DATATYPE*)malloc(size_b);
    matrix_transpose_baseline(h_a, baseline, INPUT_M, INPUT_N);
    // 分配设备上的内存
    DATATYPE* d_a = NULL;
    hipMalloc((void**)&d_a, size_a);
    DATATYPE* d_b = NULL;
    hipMalloc((void**)&d_b, size_b);
    hipError_t err = hipGetLastError();
    if (err != 0)
    {
        printf("error in hipMalloc: %s\n", hipGetErrorString(err));
    }
    // 数据拷贝
    hipMemcpy(d_a, h_a, size_a, hipMemcpyHostToDevice);
    err = hipGetLastError();
    if (err != 0)
    {
        printf("error in hipMemcpy: %s\n", hipGetErrorString(err));
    }
    {
        // 定义启动核函数的参数
        int nbx = (INPUT_N + BLOCK_DIM - 1) / BLOCK_DIM;
        int nby = (INPUT_M + BLOCK_DIM - 1) / BLOCK_DIM;
        dim3 blocksPerGrid(nbx, nby, 1);
        dim3 threadsPerBlock(BLOCK_DIM, BLOCK_DIM, 1);
        matrix_transposition_gpu_2d_2<<<blocksPerGrid, threadsPerBlock>>>(
            d_a, d_b, INPUT_M, INPUT_N);
        err = hipGetLastError();
        if (err != 0)
        {
            printf("error in kernel forward: %s\n", hipGetErrorString(err));
        }
        // 拷贝输出数据
        hipMemcpy(h_b, d_b, size_b, hipMemcpyDeviceToHost);
        check_matrix(baseline, h_b, INPUT_N, INPUT_M);
    }
    // 内存释放
    hipFree(d_a);
    hipFree(d_b);
    free(h_a);
    free(h_b);
    return 0;
}
