#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define DATATYPE float
#define BLOCKS 4
#define THREADS 4
#define INPUT_M 8
#define INPUT_N 16
#define INPUT_K 32


void print_matrix(DATATYPE* v, int m, int n)
{
    for (int i = 0; i < m; ++i)
    {
	for (int j = 0; j < n; ++j)
	{
	    printf("%f ", v[i * n + j]);
	}
	printf("\n");
    }
}


void matrix_multiplication_serial_1(DATATYPE* a, DATATYPE* b, DATATYPE* c, int m, int n, int l)
{
    double temp = 0.0;
    for (int i = 0; i < m; ++i)
    {
        for (int j = 0; j < n; ++j)
	{
	    temp = 0.0;
	    for (int k = 0; k < l; ++k)
	    {
		temp += a[i * l + k] * b[k * n + j];
	    }
	    c[i * n + j] = temp;
	}
    }
}


void matrix_multiplication_serial_2(DATATYPE* a, DATATYPE* b, DATATYPE* c, int m, int n, int l)
{
    double temp = 0.0;
    for (int i = 0; i < m * n; ++i)
    {
	c[i] = 0.0;
    }
    for (int i = 0; i < m; ++i)
    {
        for (int k = 0; k < l; ++k)
	{
	    temp = a[i * l + k];
	    for (int j = 0; j < n; ++j)
	    {
	        c[i * n + j] += temp * b[k * n + j];
	    }
	}
    }
}


void matrix_multiplication_serial_3(DATATYPE* a, DATATYPE* b, DATATYPE* c, int m, int n, int l)
{
    double temp = 0.0;
    DATATYPE* b_t = (DATATYPE*)malloc(sizeof(DATATYPE) * l * n);
    for (int i = 0; i < l; ++i)
    {
	for (int j = 0; j < n; ++j)
	{
	    b_t[i * l + j] = b[j * n + i];
	}
    }
    for (int i = 0; i < m; ++i)
    {
        for (int j = 0; j < n; ++j)
	{
	    temp = 0.0;
	    for (int k = 0; k < l; ++k)
	    {
		temp += a[i * l + k] * b_t[j * n + k];
	    }
	    c[i * n + j] = temp;
	}
    }
    free(b_t);
}


int main()
{
    int input_flag = 0;
    printf("input number to call different function: ");
    scanf("%d", &input_flag);
    srand(20);
    size_t size_a = sizeof(DATATYPE) * INPUT_M * INPUT_K;
    size_t size_b = sizeof(DATATYPE) * INPUT_K * INPUT_N;
    DATATYPE* h_a = (DATATYPE*)malloc(size_a);
    DATATYPE* h_b = (DATATYPE*)malloc(size_b);
    // initialize input vector
    for (int i = 0; i < INPUT_M; ++i)
    {
	for (int j = 0; j < INPUT_K; ++j)
	{
	    h_a[i * INPUT_K + j] = rand() / (DATATYPE)RAND_MAX;
	}
    }
    for (int i = 0; i < INPUT_K; ++i)
    {
	for (int j = 0; j < INPUT_N; ++j)
	{
	    h_a[i * INPUT_N + j] = rand() / (DATATYPE)RAND_MAX;
	}
    }
    // allocate the device for input and output
    DATATYPE* d_a = NULL;
    hipMalloc((void**)&d_a, size_a);
    DATATYPE* d_b = NULL;
    hipMalloc((void**)&d_b, size_b);
    DATATYPE* h_c = (DATATYPE*)malloc(sizeof(DATATYPE) * INPUT_M * INPUT_N);
    DATATYPE* d_c = NULL;
    hipMalloc((void**)d_c, sizeof(DATATYPE) * INPUT_M * INPUT_N);
    if (input_flag == 0)
    {
        matrix_multiplication_serial_1(h_a, h_b, h_c, INPUT_M, INPUT_N, INPUT_K);
    }
    else if (input_flag == 1)
    {
        matrix_multiplication_serial_2(h_a, h_b, h_c, INPUT_M, INPUT_N, INPUT_K);
    }
    else if (input_flag == 2)
    {
        matrix_multiplication_serial_3(h_a, h_b, h_c, INPUT_M, INPUT_N, INPUT_K);
    }
    hipMemcpy(h_c, d_c, sizeof(DATATYPE) * INPUT_M * INPUT_N, hipMemcpyDeviceToHost);
    print_matrix(h_c, INPUT_M, INPUT_N);
    hipFree(d_c);
    // memory delete
    hipFree(d_a);
    hipFree(d_b);
    return 0;
}

