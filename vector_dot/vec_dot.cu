#include "hip/hip_runtime.h"
#include <common/utils.h>


#define BLOCKS 4
#define THREADS 512
#define NUM_INPUT 2000

__device__ unsigned int lockcount = 0;


void vector_add_serial(DATATYPE* a, DATATYPE* b, DATATYPE* c)
{
    double temp = 0.0;
    for (int i = 0; i < NUM_INPUT; ++i)
    {
        temp += a[i] * b[i];
    }
    *c = temp;
}


// single block, vector reduction
__global__ void vector_dot_product_gpu_1(DATATYPE* a, DATATYPE* b, DATATYPE* c)
{
    __shared__ DATATYPE tmp[THREADS];
    const int tidx = threadIdx.x;
    const int t_n = blockDim.x;
    double temp = 0.0;
    for (int tid = tidx; tid < NUM_INPUT; tid += t_n)
    {
	temp += a[tid] * b[tid];
    }
    tmp[tidx] = temp;
    __syncthreads();
    int i = 2, j = 1;
    while (i <= THREADS)
    {
	// make the next element reduce to current even element
        if ((tidx % i) == 0)
	{
	    tmp[tidx] += tmp[tidx + j];
	}
	__syncthreads();
	i *= 2;
	j *= 2;
    }
    // all elements reduce to the first element
    if (tidx == 0)
    {
	c[0] = tmp[0];
    }
}


// single block, vector reduction
__global__ void vector_dot_product_gpu_2(DATATYPE* a, DATATYPE* b, DATATYPE* c)
{
    __shared__ DATATYPE tmp[NUM_INPUT];
    int tidx = threadIdx.x;
    const int t_n = blockDim.x;
    double temp = 0.0;
    for (int tid = tidx; tid < NUM_INPUT; tid += t_n)
    {
	temp += a[tid] * b[tid];
    }
    tmp[tidx] = temp;
    __syncthreads();
    int i = NUM_INPUT / 2;
    while (i != 0)
    {
	if (tidx < i)
	{
	    tmp[tidx] += tmp[tidx + i];
	}
	__syncthreads();
	i /= 2;
    }
    // all elements reduce to the first element
    if (tidx == 0)
    {
	c[0] = tmp[0];
    }
}


// multiple block, second reduction in cpu
__global__ void vector_dot_product_gpu_3(DATATYPE* a, DATATYPE* b, DATATYPE* c_temp)
{
    __shared__ DATATYPE tmp[NUM_INPUT];
    const int tidx = threadIdx.x;
    const int bidx = blockIdx.x;
    const int t_n = blockDim.x * gridDim.x;
    int tid = bidx * blockDim.x + tidx;
    double temp = 0.0;
    for (; tid < NUM_INPUT; tid += t_n)
    {
	temp += a[tid] * b[tid];
    }
    tmp[tidx] = temp;
    __syncthreads();
    int i = NUM_INPUT / 2;
    while (i != 0)
    {
	if (tidx < i)
	{
	    tmp[tidx] += tmp[tidx + i];
	}
	__syncthreads();
	i /= 2;
    }
    // all elements reduce to one vector which size is number of block
    if (tidx == 0)
    {
	c_temp[bidx] = tmp[0];
    }
}


// multiple block, second reduction in gpu
__global__ void vector_dot_product_gpu_4(DATATYPE* c_temp, DATATYPE* c)
{
    __shared__ DATATYPE tmp[BLOCKS];
    const int tidx = threadIdx.x;
    tmp[tidx] = c_temp[tidx];
    __syncthreads();
    int i = BLOCKS / 2;
    while (i != 0)
    {
	if (tidx < i)
	{
	    tmp[tidx] += tmp[tidx + i];
	}
	__syncthreads();
	i /= 2;
    }
    // all elements reduce to the first element
    if (tidx == 0)
    {
	c[0] = tmp[0];
    }
}


// use aotmic to reduce vector add
__global__ void vector_dot_product_gpu_5(DATATYPE* a, DATATYPE* b, DATATYPE* c)
{
    // initialize to zero
    if ((threadIdx.x == 0) && (blockIdx.x == 0))
    {
	c[0] = 0.0;
    }
    const int tidx = threadIdx.x;
    const int bidx = blockIdx.x;
    const int t_n = blockDim.x * gridDim.x;
    int tid = bidx * blockDim.x + tidx;
    double temp = 0.0;
    for (; tid < NUM_INPUT; tid += t_n)
    {
	temp += a[tid] * b[tid];
    }
    atomicAdd(c, temp);
}


// use aotmic to reduce vector add
__global__ void vector_dot_product_gpu_6(DATATYPE* a, DATATYPE* b, DATATYPE* c)
{
    // initialize to zero
    if ((threadIdx.x == 0) && (blockIdx.x == 0))
    {
	c[0] = 0.0;
    }
    __shared__ DATATYPE tmp[NUM_INPUT];
    const int tidx = threadIdx.x;
    const int bidx = blockIdx.x;
    const int t_n = blockDim.x * gridDim.x;
    int tid = bidx * blockDim.x + tidx;
    double temp = 0.0;
    for (; tid < NUM_INPUT; tid += t_n)
    {
	temp += a[tid] * b[tid];
    }
    tmp[tidx] = temp;
    __syncthreads();
    int i = blockDim.x / 2;
    while (i != 0)
    {
	if (tidx < i)
	{
	    tmp[tidx] += tmp[tidx + i];
	}
	__syncthreads();
	i /= 2;
    }
    if (tidx == 0)
    {
	atomicAdd(c, tmp[0]);
    }
}


__device__ void vector_dot(DATATYPE* out, DATATYPE* temp)
{
    const int tidx = threadIdx.x;
    int i = blockDim.x / 2;
    while (i != 0)
    {
	if (tidx < i)
	{
	    temp[tidx] += temp[tidx + i];
	}
	__syncthreads();
	i /= 2;
    }
    if (tidx == 0)
    {
	out[0] = temp[0];
    }
}


__global__ void vector_dot_product_gpu_7(DATATYPE* a, DATATYPE* b, DATATYPE* c, DATATYPE* c_temp)
{
    __shared__ DATATYPE tmp[NUM_INPUT];
    const int tidx = threadIdx.x;
    const int bidx = blockIdx.x;
    const int t_n = blockDim.x * gridDim.x;
    int tid = bidx * blockDim.x + tidx;
    double temp = 0.0;
    for (; tid < NUM_INPUT; tid += t_n)
    {
	temp += a[tid] * b[tid];
    }
    tmp[tidx] = temp;
    __syncthreads();
    vector_dot(&c_temp[blockIdx.x], tmp);
    __shared__ bool lock;
    __threadfence();
    if (tidx == 0)
    {
	unsigned int lockiii = atomicAdd(&lockcount, 1);
        lock = (lockcount == gridDim.x);
    }
    __syncthreads();
    if (lock)
    {
	tmp[tidx] = c_temp[tidx];
	__syncthreads();
	vector_dot(c, tmp);
	lockcount = 0;
    }
}


int main()
{
    int input_flag = 0;
    printf("input number to call different function: ");
    scanf("%d", &input_flag);
    srand(20);
    size_t size = sizeof(DATATYPE) * NUM_INPUT;
    DATATYPE* h_a = (DATATYPE*)malloc(size);
    DATATYPE* h_b = (DATATYPE*)malloc(size);
    // initialize input vector
    for (int i = 0; i < NUM_INPUT; ++i)
    {
	h_a[i] = rand() / (DATATYPE)RAND_MAX;
	h_b[i] = rand() / (DATATYPE)RAND_MAX;
    }
    // allocate the device for input and output
    DATATYPE* d_a = NULL;
    hipMalloc((void**)&d_a, size);
    DATATYPE* d_b = NULL;
    hipMalloc((void**)&d_b, size);
    // memory copy
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);
    if (input_flag == 0)
    {
	DATATYPE* h_c = (DATATYPE*)malloc(sizeof(DATATYPE));
        vector_add_serial(h_a, h_b, h_c);
	printf("result: %f\n", *h_c);
    }
    else if (input_flag == 1)
    {
        int threadsPerBlock = THREADS;
        int blocksPerGrid = 1;
	DATATYPE* h_c = (DATATYPE*)malloc(sizeof(DATATYPE));
        DATATYPE* d_c = NULL;
        hipMalloc((void**)&d_c, sizeof(DATATYPE));
        vector_dot_product_gpu_1<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c);
        // memory copy
        hipMemcpy(h_c, d_c, sizeof(DATATYPE), hipMemcpyDeviceToHost);
	printf("result: %f\n", *h_c);
        hipFree(d_c);
    }
    else if (input_flag == 2)
    {
        int threadsPerBlock = THREADS;
        int blocksPerGrid = 1;
	DATATYPE* h_c = (DATATYPE*)malloc(sizeof(DATATYPE));
        DATATYPE* d_c = NULL;
        hipMalloc((void**)&d_c, sizeof(DATATYPE));
        vector_dot_product_gpu_2<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c);
        // memory copy
        hipMemcpy(h_c, d_c, sizeof(DATATYPE), hipMemcpyDeviceToHost);
	printf("result: %f\n", *h_c);
        hipFree(d_c);
    }
    else if (input_flag == 3)
    {
        int threadsPerBlock = THREADS;
        int blocksPerGrid = BLOCKS;
	int blocknum = blocksPerGrid;
	DATATYPE* h_c = (DATATYPE*)malloc(sizeof(DATATYPE) * blocknum);
        DATATYPE* d_c = NULL;
        hipMalloc((void**)&d_c, sizeof(DATATYPE) * blocknum);
        vector_dot_product_gpu_3<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c);
        // memory copy
        hipMemcpy(h_c, d_c, sizeof(DATATYPE) * blocknum, hipMemcpyDeviceToHost);
	// vector reduction in cpu
	DATATYPE temp = 0.0;
	for (int i = 0; i < blocknum; ++i)
	{
	    temp += h_c[i];
	}
        printf("result: %f\n", temp);
        hipFree(d_c);
    }
    else if (input_flag == 4)
    {
        int threadsPerBlock = THREADS;
        int blocksPerGrid = BLOCKS;
	int blocknum = blocksPerGrid;
	DATATYPE* h_c = (DATATYPE*)malloc(sizeof(DATATYPE));
        DATATYPE* d_c_temp = NULL;
        hipMalloc((void**)&d_c_temp, sizeof(DATATYPE) * blocknum);
        vector_dot_product_gpu_3<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c_temp);
	// vector reduction in gpu
	DATATYPE* d_c = NULL;
        hipMalloc((void**)&d_c, sizeof(DATATYPE));
        vector_dot_product_gpu_4<<<1, blocksPerGrid>>>(d_c_temp, d_c);
        // memory copy
        hipMemcpy(h_c, d_c, sizeof(DATATYPE), hipMemcpyDeviceToHost);
        printf("result: %f\n", *h_c);
        hipFree(d_c);
    }
    else if (input_flag == 5)
    {
        int threadsPerBlock = THREADS;
        int blocksPerGrid = BLOCKS;
	DATATYPE* h_c = (DATATYPE*)malloc(sizeof(DATATYPE));
        DATATYPE* d_c = NULL;
        hipMalloc((void**)&d_c, sizeof(DATATYPE));
        vector_dot_product_gpu_5<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c);
        // memory copy
        hipMemcpy(h_c, d_c, sizeof(DATATYPE), hipMemcpyDeviceToHost);
	// vector reduction in cpu
        printf("result: %f\n", *h_c);
        hipFree(d_c);
    }
    else if (input_flag == 6)
    {
        int threadsPerBlock = THREADS;
        int blocksPerGrid = BLOCKS;
	DATATYPE* h_c = (DATATYPE*)malloc(sizeof(DATATYPE));
        DATATYPE* d_c = NULL;
        hipMalloc((void**)&d_c, sizeof(DATATYPE));
        vector_dot_product_gpu_6<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c);
        // memory copy
        hipMemcpy(h_c, d_c, sizeof(DATATYPE), hipMemcpyDeviceToHost);
	// vector reduction in cpu
        printf("result: %f\n", *h_c);
        hipFree(d_c);
    }
    else if (input_flag == 7)
    {
        int threadsPerBlock = THREADS;
        int blocksPerGrid = BLOCKS;
	int blocknum = blocksPerGrid;
	DATATYPE* h_c = (DATATYPE*)malloc(sizeof(DATATYPE));
        DATATYPE* d_c = NULL;
        hipMalloc((void**)&d_c, sizeof(DATATYPE));
        DATATYPE* d_c_temp = NULL;
        hipMalloc((void**)&d_c_temp, sizeof(DATATYPE) * blocknum);
        vector_dot_product_gpu_7<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, d_c_temp);
        // memory copy
        hipMemcpy(h_c, d_c, sizeof(DATATYPE), hipMemcpyDeviceToHost);
	// vector reduction in cpu
        printf("result: %f\n", *h_c);
        hipFree(d_c);
    }
    // memory delete
    hipFree(d_a);
    hipFree(d_b);
    // return
    return 0;
}
